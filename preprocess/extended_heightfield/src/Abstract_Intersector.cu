#include "hip/hip_runtime.h"
#include "Abstract_Intersector.h"

#include "cuda_utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <string>

template<class Primitive>
Abstract_Intersector<Primitive>::Abstract_Intersector( std::tuple<int, int> output_resolution, int n_hf_entries, int buffer_length )
	: output_resolution( as_int2(output_resolution) )
	, n_hf_entries(n_hf_entries)
	, buffer_length(buffer_length)
{
	extended_heightfield = new GPUMappedFloat2Buffer( make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), buffer_length), empty_interval );
	normal_map = new GPUMappedFloat3Buffer( make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), 1) );
	z_buffer = new GPUMappedFloatBuffer( make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), 1) );
}

template<class Primitive>
Abstract_Intersector<Primitive>::Abstract_Intersector(float2* extended_heightfield_gpu, float* z_buffer_gpu, float3* normal_map_gpu, std::tuple<int, int> output_resolution, int n_hf_entries, int buffer_length)
	: output_resolution( as_int2(output_resolution ) )
	, n_hf_entries(n_hf_entries)
	, buffer_length(buffer_length)
{
	extended_heightfield = new GPUMappedFloat2Buffer(make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), buffer_length), extended_heightfield_gpu);
	normal_map = new GPUMappedFloat3Buffer(make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), 1), normal_map_gpu);
	z_buffer = new GPUMappedFloatBuffer(make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), 1), z_buffer_gpu);
}

template<class Primitive>
void Abstract_Intersector<Primitive>::add_primitives(std::vector<Primitive>& primitives)
{
	primitives_cpu = primitives;
	n_primitives = (int)primitives.size();
	presort_primitives();
	primitives_gpu = allocate_primitives_on_gpu(primitives_cpu);
}

template<class Primitive>
void Abstract_Intersector<Primitive>::add_primitives_py(py::array& primitives)
{
	allocate_primitives_cpu(primitives);
	presort_primitives();
	primitives_gpu = allocate_primitives_on_gpu(primitives_cpu);
}

template<class Primitive>
Abstract_Intersector<Primitive>::~Abstract_Intersector<Primitive>()
{
	delete( extended_heightfield );
	delete( normal_map );
}

template<class Primitive>
std::tuple< py::array_t<float>, py::array_t<float> >  Abstract_Intersector<Primitive>::intersect_py( float image_plane )
{
	intersect( image_plane );
	return std::tuple<py::array_t<float>, py::array_t<float> >(get_extended_height_field_py(), get_normal_map_py());
}

template<class Primitive>
py::array_t<float3> Abstract_Intersector<Primitive>::get_normal_map_py()
{
	normal_map->pull_from_gpu();
	return normal_map->as_py();
}

template<class Primitive>
float3* Abstract_Intersector<Primitive>::get_normal_map()
{
	normal_map->pull_from_gpu();
	return normal_map->cpu_ptr();
}

template<class Primitive>
py::array_t<float> Abstract_Intersector<Primitive>::get_extended_height_field_py()
{
	extended_heightfield->pull_from_gpu();
	return extended_heightfield->as_py();
}

template<class Primitive>
void Abstract_Intersector<Primitive>::allocate_primitives_cpu(py::array& primitives)
{
	py::buffer_info info = primitives.request();
	if (info.ndim != 2)
		throw std::invalid_argument("primitives array is expected to be of two dimensions, found "+std::to_string(info.ndim));
	if (info.shape[1] != Primitive::N_FLOAT_PARAMS)
		throw std::invalid_argument("primitives array is expected to be of dimensions nx" + std::to_string(Primitive::N_FLOAT_PARAMS) + ", found " + std::to_string(info.shape[0]) + "x" + std::to_string(info.shape[1]) );
	if (info.format != "f")
		throw std::invalid_argument("spheres array is expected to be of dtype float32, found " + info.format);
	n_primitives = info.shape[0];
	primitives_cpu.resize(n_primitives);
	float* ptr = (float*) info.ptr;
	for (size_t i = 0; i < n_primitives; i++)
	{
		primitives_cpu[i] = ptr;
		ptr += Primitive::N_FLOAT_PARAMS;
	}
}

template<class Primitive>
Primitive* Abstract_Intersector<Primitive>::allocate_primitives_on_gpu( const std::vector<Primitive>& primitives_cpu )
{
	Primitive* ptr_gpu;
	hipMalloc((void**)&ptr_gpu, sizeof(Primitive) * n_primitives);
	hipMemcpy(ptr_gpu, &primitives_cpu[0], sizeof(Primitive) * n_primitives, hipMemcpyHostToDevice);
	return ptr_gpu;
}

template<class Primitive>
void Abstract_Intersector<Primitive>::presort_primitives()
{
	assign_aabb();
	if (primitives_cpu.size() == 0)
		throw std::runtime_error("no primitives in call to presort");
	std::sort(primitives_cpu.begin(), primitives_cpu.end(), primitives_cpu[0]);
}

#include "sphere.h"
#include "cylinder.h"
#include "cuboid.h"
template class Abstract_Intersector<Sphere>;
template class Abstract_Intersector<Cylinder>;
template class Abstract_Intersector<Cuboid>;